#include "hip/hip_runtime.h"
__global__ void single_query_cached_kv_attention_kernel(
  scalar_t* __restrict__ out,             // [num_seqs, num_heads, head_size]
  const scalar_t* __restrict__ q,         // [num_seqs, num_heads, head_size]
  const scalar_t* __restrict__ k_cache,   // [num_blocks, num_kv_heads, head_size/x, block_size, x]，最后一个x是vectorize，一个thread fetch一个vector
  const scalar_t* __restrict__ v_cache,   // [num_blocks, num_kv_heads, head_size, block_size], num_blocks * block_size=seqlen
  const int* __restrict__ head_mapping,   // [num_heads]，q与kv的head map
  const float scale,
  const int* __restrict__ block_tables,   // [num_seqs, max_num_blocks_per_seq],2d数组，每个子数组是每个seq的存储kv的physical block nums
  const int* __restrict__ context_lens,   // [num_seqs]，每个句子的长度
  const int max_num_blocks_per_seq, //(max(context_lens) + block_size - 1) / block_size 
  const float* __restrict__ alibi_slopes, // [num_heads]
  const int q_stride,
  const int kv_block_stride,//类似于pytorch的stride，每个physical block的stride
  const int kv_head_stride) //类似于pytorch的stride，每个head的stride
  // $ head_size 就是qkv的embedding_size
  constexpr int THREAD_GROUP_SIZE = MAX(WARP_SIZE / BLOCK_SIZE, 1);// 每个thread_group 处理blocksize中的1个token，每个token又有numheads * headsize个element，每个block有block size个token，
  constexpr int NUM_THREAD_GROUPS = NUM_THREADS / THREAD_GROUP_SIZE; // Note: This assumes THREAD_GROUP_SIZE divides NUM_THREADS
  assert(NUM_THREADS % THREAD_GROUP_SIZE == 0);
  //每组thread处理的token数量，最小为1
  constexpr int NUM_TOKENS_PER_THREAD_GROUP = (BLOCK_SIZE + WARP_SIZE - 1) / WARP_SIZE;
  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  const int thread_idx = threadIdx.x;
  const int warp_idx = thread_idx / WARP_SIZE;
  const int lane = thread_idx % WARP_SIZE;

  const int head_idx = blockIdx.x;   //一个block负责一个head，headsize*blocksize的数据
  const int num_heads = gridDim.x;//
  const int kv_head_idx = head_mapping[head_idx]; // q head id->kv head id
  const int seq_idx = blockIdx.y ; // y维度的一个block负责一个seq

  // 每个thread group 向量化load&store，这里其实我有点疑问，为什么是以thread group为单位load 16*8=128bit数据，而不是以thread，因为CUDA每个thread一次性最大可以访问128bit数据
  constexpr int VEC_SIZE = MAX(16 / (THREAD_GROUP_SIZE * sizeof(scalar_t)), 1); 
  using K_vec = typename Vec<scalar_t, VEC_SIZE>::Type;
  using Q_vec = typename Vec<scalar_t, VEC_SIZE>::Type;
  // 1个thread group处理一个head里面的head size
  constexpr int NUM_ELEMS_PER_THREAD = HEAD_SIZE / THREAD_GROUP_SIZE;
  constexpr int NUM_VECS_PER_THREAD = NUM_ELEMS_PER_THREAD / VEC_SIZE;
  // 当前thread所在的thread group
  const int thread_group_idx = thread_idx / THREAD_GROUP_SIZE;
  // 当前thread在thread_group内的offset
  const int thread_group_offset = thread_idx % THREAD_GROUP_SIZE;


    const scalar_t* q_ptr = q + seq_idx * q_stride + head_idx * HEAD_SIZE;
  //每个block x负责一个head，那么这里申请一块shared mem来存每个thread x读到的head size维度数据
  __shared__ Q_vec q_vecs[THREAD_GROUP_SIZE][NUM_VECS_PER_THREAD];
  for (int i = thread_group_idx; i < NUM_VECS_PER_THREAD; i += NUM_THREAD_GROUPS) {
    const int vec_idx = thread_group_offset + i * THREAD_GROUP_SIZE;
    // 每个thread读取的q vector都放在q_vecs, 求出当前thread处理的q的最后一维的offset=q_ptr + vec_idx * VEC_SIZE
    q_vecs[thread_group_offset][i] = *reinterpret_cast<const Q_vec*>(q_ptr + vec_idx * VEC_SIZE);
  }